#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
__constant__ float T, r, sigma, rho, alpha, dt, con1, con2;


////////////////////////////////////////////////////////////////////////
// kernel routines -- see sections 3.5, 3.6 in cuRAND documentation
////////////////////////////////////////////////////////////////////////

__global__ void RNG_init(hiprandState *state)
{
  // RNG initialisation with id-based skipahead
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  hiprand_init(1234, id, 0, &state[id]);
}


__global__ void pathcalc(hiprandState *device_state, float *d_v,
                         int mpath, int NPATH)
{
  float s1, s2, y1, y2, payoff;

  int id = threadIdx.x + blockIdx.x*blockDim.x;
  hiprandState_t state = device_state[id];

  for(int m=0; m<mpath; m++) {
    s1 = 1.0f;
    s2 = 1.0f;

    for (int n=0; n<N; n++) {
      y1 = hiprand_normal(&state);
      y2 = rho*y1 + alpha*hiprand_normal(&state);

      s1 = s1*(con1 + con2*y1);
      s2 = s2*(con1 + con2*y2);
    }

    // put payoff value into device array

    payoff = 0.0f;
    if ( fabs(s1-1.0f)<0.1f && fabs(s2-1.0f)<0.1f ) payoff = exp(-r*T);

    int payoff_id = id + m*gridDim.x*blockDim.x;
    if (payoff_id < NPATH) d_v[payoff_id] = payoff;
  }
}


////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){
    
  int     NPATH=9600000, h_N=100;
  float   h_T, h_r, h_sigma, h_rho, h_alpha, h_dt, h_con1, h_con2;
  float  *h_v, *d_v;
  double  sum1, sum2;
  hiprandState *state;

  // initialise card

  findCudaDevice(argc, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // allocate memory on host and device

  h_v = (float *)malloc(sizeof(float)*NPATH);
  checkCudaErrors( hipMalloc((void **)&d_v, sizeof(float)*NPATH) );
  checkCudaErrors( hipMalloc((void **)&state, sizeof(hiprandState)*NPATH) );

  printf("size of hiprandState is %d bytes\n",sizeof(hiprandState));

  // define constants and transfer to GPU

  h_T     = 1.0f;
  h_r     = 0.05f;
  h_sigma = 0.1f;
  h_rho   = 0.5f;
  h_alpha = sqrt(1.0f-h_rho*h_rho);
  h_dt    = 1.0f/h_N;
  h_con1  = 1.0f + h_r*h_dt;
  h_con2  = sqrt(h_dt)*h_sigma;

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(T),    &h_T,    sizeof(h_T)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(r),    &h_r,    sizeof(h_r)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sigma),&h_sigma,sizeof(h_sigma)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(rho),  &h_rho,  sizeof(h_rho)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(alpha),&h_alpha,sizeof(h_alpha)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dt),   &h_dt,   sizeof(h_dt)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con1), &h_con1, sizeof(h_con1)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con2), &h_con2, sizeof(h_con2)) );

  // calculate theoretical occupancy -- see Pro Tip blog article:
  // https://developer.nvidia.com/blog/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/

  int device;
  hipDeviceProp_t props;
  hipGetDevice(&device);
  hipGetDeviceProperties(&props, device);
  
  int maxActiveBlocks, blockSize=128;
  hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, 
                                                 pathcalc, blockSize, 0);
  printf("maxActiveBlocks/SM = %d \n",maxActiveBlocks);
  printf("number of SMs      = %d \n",props.multiProcessorCount);
  int blocks = maxActiveBlocks*props.multiProcessorCount;
  
  // execute kernels

  hipEventRecord(start);
  RNG_init<<<blocks, 128>>>(state);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  getLastCudaError("RNG_init execution failed\n");
  printf("RNG_init kernel execution time (ms): %f \n",milli);

  int paths_per_thread = (NPATH-1)/(128*blocks) + 1;
  hipEventRecord(start);
  pathcalc<<<blocks, 128>>>(state,d_v,paths_per_thread,NPATH);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  getLastCudaError("pathcalc execution failed\n");
  printf("pathcalc kernel execution time (ms): %f \n",milli);

  // copy back results

  checkCudaErrors( hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
                   hipMemcpyDeviceToHost) );

  // compute average

  sum1 = 0.0;
  sum2 = 0.0;
  for (int i=0; i<NPATH; i++) {
    sum1 += h_v[i];
    sum2 += h_v[i]*h_v[i];
  }

  printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
	 sum1/NPATH, sqrt((sum2/NPATH - (sum1/NPATH)*(sum1/NPATH))/NPATH) );

  // Release memory and exit cleanly

  free(h_v);
  checkCudaErrors( hipFree(d_v) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

}
