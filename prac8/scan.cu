#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

#include <hip/hip_runtime_api.h>


///////////////////////////////////////////////////////////////////////
// CPU routine
///////////////////////////////////////////////////////////////////////

void scan_gold(float* odata, float* idata, const unsigned int len) 
{
  odata[0] = 0;
  for(int i=1; i<len; i++) odata[i] = idata[i-1] + odata[i-1];
}

///////////////////////////////////////////////////////////////////////
// GPU routine
///////////////////////////////////////////////////////////////////////

__global__ void scan(float *g_odata, float *g_idata)
{
  // Dynamically allocated shared memory for scan kernels

  extern __shared__  float tmp[];

  float temp;
  int   tid = threadIdx.x;

  // read input into shared memory

  temp     = g_idata[tid];
  tmp[tid] = temp;

  // perform scan

  for (int d=1; d<blockDim.x; d=2*d) {
    __syncthreads();
    if (tid-d >= 0) temp += tmp[tid-d];
    __syncthreads();
    tmp[tid] = temp;
  }

  // write results to global memory

  __syncthreads();

  temp = 0.0f;
  if (tid>0) temp = tmp[tid-1];

  g_odata[tid] = temp;
}


////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////

int main( int argc, const char** argv) 
{
  int num_threads, num_elements, mem_size, shared_mem_size;

  float *h_data, *reference;
  float *d_idata, *d_odata;

  // initialise card

  findCudaDevice(argc, argv);

  num_threads  = 512;
  num_elements = num_threads;
  mem_size     = sizeof(float) * num_elements;

  // allocate host memory to store the input data
  // and initialize to integer values between 0 and 10

  h_data = (float*) malloc(mem_size);
      
  for(int i=0; i<num_elements; i++) 
    h_data[i] = floorf(10.0f*(rand()/(float)RAND_MAX));

  // compute reference solution

  reference = (float*) malloc(mem_size);
  scan_gold(reference, h_data, num_elements);

  // allocate device memory input and output arrays

  checkCudaErrors( hipMalloc((void**)&d_idata, mem_size) );
  checkCudaErrors( hipMalloc((void**)&d_odata, mem_size) );

  // copy host memory to device input array

  checkCudaErrors( hipMemcpy(d_idata, h_data, mem_size,
                              hipMemcpyHostToDevice));

  // execute the kernel

  shared_mem_size = sizeof(float) * num_threads;
  scan<<<1,num_threads,shared_mem_size>>>(d_odata,d_idata);
  getLastCudaError("scan kernel execution failed");

  // copy result from device to host

  checkCudaErrors( hipMemcpy(h_data, d_odata, mem_size,
                              hipMemcpyDeviceToHost) );

  // check results

  float err=0.0;
  for (int i=0; i<num_elements; i++)
    err += (h_data[i] - reference[i])*(h_data[i] - reference[i]);
  printf("rms scan error  = %f\n",sqrt(err/num_elements));

  // cleanup memory

  free(h_data);
  free(reference);
  checkCudaErrors( hipFree(d_idata) );
  checkCudaErrors( hipFree(d_odata) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();
}
